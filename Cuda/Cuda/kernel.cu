#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define MATRIX_SIZE 2048

using namespace std;

__global__ void matrixMultiply(double* matA, double* matB, double* resultMat) {
    double sum;
    int rowStart = blockIdx.x, blockCount = gridDim.x;
    int colStart = threadIdx.x, threadCount = blockDim.x;

    for (int i = rowStart; i < MATRIX_SIZE; i += blockCount)
        for (int j = colStart; j < MATRIX_SIZE; j += threadCount) {
            sum = 0;
            for (int k = 0; k < MATRIX_SIZE; ++k)
                sum += matA[i * MATRIX_SIZE + k] * matB[k * MATRIX_SIZE + j];
            resultMat[i * MATRIX_SIZE + j] = sum;
        }
}

int main() {
    double* matA, * matB, * resultMat;
    int memSize = MATRIX_SIZE * MATRIX_SIZE * sizeof(double);
    hipEvent_t eventStart, eventStop;
    float elapsedTime;

    hipMallocManaged(&matA, memSize);
    hipMallocManaged(&matB, memSize);
    hipMallocManaged(&resultMat, memSize);

    for (int i = 0; i < MATRIX_SIZE * MATRIX_SIZE; ++i)
        matA[i] = matB[i] = 2;

    int blockCount = 32, threadCount = 1024;
    dim3 threads(threadCount);
    dim3 blocks(blockCount);

    hipEventCreate(&eventStart);
    hipEventCreate(&eventStop);
    hipEventRecord(eventStart, 0);

    matrixMultiply << <blocks, threads >> > (matA, matB, resultMat);
    hipDeviceSynchronize();

    hipEventRecord(eventStop, 0);
    hipEventSynchronize(eventStop);
    hipEventElapsedTime(&elapsedTime, eventStart, eventStop);
    printf("Block count = %i, Thread count = %i, Time = %f ms\n", blockCount, threadCount, elapsedTime);

    hipEventDestroy(eventStart);
    hipEventDestroy(eventStop);
    hipFree(matA);
    hipFree(matB);
    hipFree(resultMat);

    return 0;
}